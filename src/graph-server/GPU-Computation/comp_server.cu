#include <cmath>
#include <iostream>

#include "../../common/utils.hpp"
#include "comp_server.cuh"
using namespace std;

void loadWeightServers(std::vector<char *> &addresses,
                       const std::string &wServersFile) {
    std::ifstream infile(wServersFile);
    if (!infile.good())
        printf("Cannot open weight server file: %s [Reason: %s]\n",
               wServersFile.c_str(), std::strerror(errno));

    assert(infile.good());

    std::string line;
    while (!infile.eof()) {
        std::getline(infile, line);
        boost::algorithm::trim(line);

        if (line.length() == 0) continue;

        char *addr = strdup(line.c_str());
        addresses.push_back(addr);
    }
}
// ComputingServer::ComputingServer() : cu(ComputingUnit::getInstance()){};

ComputingServer::ComputingServer(GPUComm *gpu_comm)
    : cu(ComputingUnit::getInstance()),
      msgService(gpu_comm->wPort, gpu_comm->nodeId) {
    gpuComm = gpu_comm;
    totalLayers = gpu_comm->totalLayers;
    nodeId = gpu_comm->nodeId;
    loadWeightServers(weightServerAddrs, gpu_comm->wServersFile);
    msgService.setUpWeightSocket(
        weightServerAddrs.at(nodeId % weightServerAddrs.size()));

    msgService.prefetchWeightsMatrix(totalLayers);
}

// Start listening to main thread
void ComputingServer::terminate() {
    // msgService.terminateWeightServers(weightServerAddrs);
}

// Start GAT-Specific Code
void ComputingServer::vtxNNForward(unsigned layer, bool lastLayer) {
    Matrix feats = (*gpuComm->tensorMap)["h"];
    Matrix weight = msgService.getWeightMatrix(layer);
    auto z = cu.dot(feats, weight);
    memcpy((*gpuComm->tensorMap)["z"].getData(), z.getData(), z.getDataSize());
    delete[] z.getData();
    CuMatrix::freeGPU();
}

void ComputingServer::edgNNForward(unsigned layer, bool lastLayer) {
    cout << "Layer " << layer << endl;
    CuMatrix *adj =
        (CuMatrix *)gpuComm->engine->adjIn;  // any engineer with pursuit should
                                             // not write this;too ugly
    CuMatrix e = *adj;
    int nnz = adj->nnz;
    CuMatrix z = cu.wrapMatrix((*gpuComm->tensorMap)["z"]);
    auto a = cu.wrapMatrix(msgService.getaMatrix(layer));
    auto az = z.dot(a);
    auto act_az = cu.leakyRelu(az, 0.01);
    az.setData((*gpuComm->tensorMap)["az"].getData());
    az.updateMatrixFromGPU();
    CuMatrix e_dst = cu.wrapMatrix(Matrix(1, nnz, (char *)NULL));
    auto cusparseStat = hipsparseSgthr(
        cu.spHandle, nnz, act_az.devPtr, e_dst.devPtr, adj->csrRowInd,
        HIPSPARSE_INDEX_BASE_ZERO);  // gather the 1st half of az
    assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);
    e_dst.setData((*gpuComm->tensorMap)["A"].getData());
    e_dst.updateMatrixFromGPU();

    CuMatrix::freeGPU();
}

void ComputingServer::edgNNBackward(unsigned layer) {
    CuMatrix a = cu.wrapMatrix(msgService.getaMatrix(layer));

    auto gradTensor = cu.wrapMatrix((*gpuComm->tensorMap)["grad"]);
    auto zaTensor = cu.wrapMatrix((*gpuComm->tensorMap)["az"]);
    unsigned featDim = gradTensor.getCols();
    CuMatrix *adj =
        (CuMatrix *)gpuComm->engine->adjIn;  // any engineer with pursuit should
                                             // not write this;too ugly
    CuMatrix e = *adj;
    unsigned edgCnt = e.nnz;

    cout<<"leakyReluPrime\n";
    CuMatrix d_lrelu = cu.leakyReluPrime(zaTensor, 0.01);  // n x 1
    zaTensor.explicitFree();
    CuMatrix d_lrelu_edge =
        cu.wrapMatrix(Matrix(e.nnz, 1, (char *)NULL));  // BCAST |V| to |E|
    cout<<"hipsparseSgthr\n";
    auto cusparseStat = hipsparseSgthr(
        cu.spHandle, e.nnz, d_lrelu.devPtr, d_lrelu_edge.devPtr,
        e.csrRowInd,  // Not sure need to see the actually adjmatrix***
        HIPSPARSE_INDEX_BASE_ZERO);  // gather the 1st half of az//
    assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);

    vector<int> src_indices(e.nnz);
    vector<int> dst_indices(e.nnz);
    hipMemcpy(src_indices.data(), e.csrRowInd, sizeof(int) * e.nnz,
               hipMemcpyDeviceToHost);
    hipMemcpy(dst_indices.data(), e.csrColInd, sizeof(int) * e.nnz,
               hipMemcpyDeviceToHost);

    cout<<"gatherRows d_P_edge \n";
    // auto d_P_edge = cu.gatherRows(gradTensor, src_indices);
    auto d_P_edge = cu.gatherRowsGthr(gradTensor, e.csrRowInd,e.nnz);
    hipDeviceSynchronize();
    cout<<"scaleRowsByVector\n";
    cu.scaleRowsByVector(d_P_edge, d_lrelu_edge);  //(|E|, featDim)
    auto d_Act = d_P_edge;
    d_lrelu.explicitFree();
    cout<<d_Act.shape()<<endl;

    if (layer != 0) {
        // Shape of dA: (|E|, 1), serve as gradient of each edge for backward
        // agg
        cout<<"d_Act.dot(a)\n";
        auto dA = d_Act.dot(a);
        dA.setData((*gpuComm->tensorMap)["dA"].getData());
        dA.updateMatrixFromGPU();
        dA.explicitFree();
    }
    cout<<"d_Act_reduce\n";
    auto d_Act_reduce = cu.reduceColumns(d_Act);
    CuMatrix::MemoryPool.erase(d_Act.devPtr);
    hipFree(d_Act.devPtr);

    cout<<"gatherRows gatherRows\n";
    auto z = cu.wrapMatrix((*gpuComm->tensorMap)["z"]);
    // auto z_src=cu.gatherRows(z, src_indices);
    // auto z_dst=cu.gatherRows(z, dst_indices);
    auto z_src=cu.gatherRowsGthr(z, e.csrRowInd,e.nnz);
    auto z_dst=cu.gatherRowsGthr(z, e.csrColInd,e.nnz);
    CuMatrix::MemoryPool.erase(z.devPtr);
    hipDeviceSynchronize();
    hipFree(z.devPtr);
    cout<<"zz=z_src.dot(z_dst\n";
    auto zz=z_src.dot(z_dst,true,false);
    cout<<"da\n";
    CuMatrix da = zz.dot(d_Act_reduce, false, true);
    da.updateMatrixFromGPU();
    msgService.sendaUpdate(da, layer);
    CuMatrix::freeGPU();
}
void ComputingServer::vtxNNBackward(unsigned layer) {
    auto weight = cu.wrapMatrix(msgService.getWeightMatrix(layer));
    auto grad = cu.wrapMatrix((*gpuComm->tensorMap)["aTg"]);
    auto h = cu.wrapMatrix((*gpuComm->tensorMap)["h"]);
    auto weightUpdates = h.dot(grad, true, false);
    weightUpdates.updateMatrixFromGPU();
    msgService.sendWeightUpdate(weightUpdates, layer);
    weightUpdates.free();

    if (layer != 0) {
        auto resultGrad = grad.dot(weight, false, true);
        resultGrad.setData(
            gpuComm->engine->savedNNTensors[layer - 1]["grad"].getData());
        resultGrad.updateMatrixFromGPU();
        printLog(
            nodeId, "layer %u, resultG %s, output %s", layer,
            resultGrad.shape().c_str(),
            gpuComm->engine->savedNNTensors[layer - 1]["grad"].shape().c_str());
        resultGrad.free();
    }

    CuMatrix::freeGPU();
}

// end GAT

void ComputingServer::processForward(unsigned layer, bool lastLayer) {
    if (layer == 0) CuMatrix::freeGPU();

    Matrix feats = (*gpuComm->tensorMap)["ah"];
    Matrix weight = msgService.getWeightMatrix(layer);
    CuMatrix z = cu.dot(feats, weight);

    if (!lastLayer) {
        Matrix savedTensor = (*gpuComm->tensorMap)["z"];
        Matrix outputTensor = (*gpuComm->tensorMap)["h"];
        FeatType *act_z = outputTensor.getData();
        FeatType *z_data = savedTensor.getData();
        memcpy(z_data, z.getData(), z.getDataSize());
        cu.activate(z);  // z data get activated ...
        z.updateMatrixFromGPU();
        memcpy(act_z, z.getData(), z.getDataSize());

    } else {  // do the last layer + the bp with it
        CuMatrix cuPredictions = cu.softmaxRows(z);
        gradLoss(layer, cuPredictions);
    }
    delete[] z.getData();
}

void ComputingServer::processBackward(unsigned layer) {
    gradLayer(layer);
    if (layer == 0) msgService.prefetchWeightsMatrix(totalLayers);
}

void ComputingServer::gradLayer(unsigned layer) {
    Matrix grad = (*gpuComm->tensorMap)["aTg"];
    CuMatrix cuGrad = cu.wrapMatrix(grad);
    Matrix z = (*gpuComm->tensorMap)["z"];
    CuMatrix cuZ = cu.wrapMatrix(z);
    Matrix ah = (*gpuComm->tensorMap)["ah"];
    CuMatrix cuAh = cu.wrapMatrix(ah);

    CuMatrix interGrad = cu.activateBackward(cuAh, cuZ, cuGrad);
    CuMatrix cuWeightUpdates = cuAh.dot(interGrad, true, false);

    Matrix weightUpdates = cuWeightUpdates.getMatrix();
    Matrix weight = msgService.getWeightMatrix(layer);
    CuMatrix cuWeights = cu.wrapMatrix(weight);
    if (layer != 0) {
        CuMatrix resultGrad = interGrad.dot(cuWeights, false, true);
        resultGrad.setData((*gpuComm->tensorMap)["grad"].getData());
        resultGrad.updateMatrixFromGPU();
    }

    msgService.sendWeightUpdate(weightUpdates, layer);
}

void ComputingServer::gradLoss(unsigned layer, CuMatrix pred, bool report) {
    // here it can be optimized by fetching directly from Forward;
    Matrix labels = (*gpuComm->tensorMap)["lab"];
    CuMatrix cuLabels = cu.wrapMatrix(labels);
    CuMatrix d_output = cu.hadamardSub(pred, cuLabels);

    if (report) {
        float acc, loss;
        cu.getTrainStat(pred, cuLabels, acc, loss);
        printLog(nodeId, "batch Acc: %f, Loss: %f\n", acc, loss);
    }

    Matrix weight = msgService.getWeightMatrix(layer);
    CuMatrix cuWeights = cu.wrapMatrix(weight);
    CuMatrix interGrad = d_output.dot(cuWeights, false, true);
    interGrad.setData((*gpuComm->tensorMap)["grad"].getData());
    interGrad.updateMatrixFromGPU();

    Matrix ah = (*gpuComm->tensorMap)["ah"];
    CuMatrix cuAh = cu.wrapMatrix(ah);
    CuMatrix cuWeightUpdates = cuAh.dot(d_output, true, false);
    Matrix weightUpdates = cuWeightUpdates.getMatrix();
    msgService.sendWeightUpdate(weightUpdates, layer);
}
