#include <cmath>
#include <iostream>

#include "../../common/utils.hpp"
#include "comp_server.cuh"
using namespace std;

void loadWeightServers(std::vector<char *> &addresses,
                       const std::string &wServersFile) {
    std::ifstream infile(wServersFile);
    if (!infile.good())
        printf("Cannot open weight server file: %s [Reason: %s]\n",
               wServersFile.c_str(), std::strerror(errno));

    assert(infile.good());

    std::string line;
    while (!infile.eof()) {
        std::getline(infile, line);
        boost::algorithm::trim(line);

        if (line.length() == 0) continue;

        char *addr = strdup(line.c_str());
        addresses.push_back(addr);
    }
}
// ComputingServer::ComputingServer() : cu(ComputingUnit::getInstance()){};

ComputingServer::ComputingServer(GPUComm *gpu_comm)
    : cu(ComputingUnit::getInstance()),
      msgService(gpu_comm->wPort, gpu_comm->nodeId) {
    gpuComm = gpu_comm;
    totalLayers = gpu_comm->totalLayers;
    nodeId = gpu_comm->nodeId;
    loadWeightServers(weightServerAddrs, gpu_comm->wServersFile);
    msgService.setUpWeightSocket(
        weightServerAddrs.at(nodeId % weightServerAddrs.size()));

    msgService.prefetchWeightsMatrix(totalLayers);
}

// Start listening to main thread
void ComputingServer::terminate() {
    // msgService.terminateWeightServers(weightServerAddrs);
}

// Start GAT-Specific Code
void ComputingServer::vtxNNForward(unsigned layer, bool lastLayer) {
    Matrix feats = (*gpuComm->tensorMap)["h"];
    Matrix h = layer == 0 ? (*gpuComm->tensorMap)["h"]
                          : gpuComm->engine->savedNNTensors[layer - 1]["ah"];
    Matrix weight = msgService.getWeightMatrix(layer);
    auto z = cu.dot(h, weight);
    memcpy((*gpuComm->tensorMap)["z"].getData(), z.getData(), z.getDataSize());
    delete[] z.getData();
    CuMatrix::freeGPU();
}

void ComputingServer::edgNNForward(unsigned layer, bool lastLayer) {
    cout << "Layer " << layer << endl;
    CuMatrix *adj =
        (CuMatrix *)gpuComm->engine->adjIn;  // any engineer with pursuit should
                                             // not write this;too ugly
    CuMatrix e = *adj;
    int nnz = adj->nnz;
    CuMatrix z = cu.wrapMatrix((*gpuComm->tensorMap)["z"]);
    auto a = cu.wrapMatrix(msgService.getaMatrix(layer));
    auto az = z.dot(a);
    az.setData((*gpuComm->tensorMap)["az"].getData());
    az.updateMatrixFromGPU();
    CuMatrix e_dst = cu.wrapMatrix(Matrix(1, nnz, (char *)NULL));
    auto cusparseStat = hipsparseSgthr(
        cu.spHandle, nnz, az.devPtr, e_dst.devPtr, adj->csrRowInd,
        HIPSPARSE_INDEX_BASE_ZERO);  
    assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);

    auto act_edge = cu.leakyRelu(e_dst, 0.01);
    e_dst.explicitFree();
    act_edge.setData((*gpuComm->tensorMap)["A"].getData());
    act_edge.updateMatrixFromGPU();
    CuMatrix::freeGPU();
}

void ComputingServer::edgNNBackward(unsigned layer) {
    auto zaTensor = cu.wrapMatrix((*gpuComm->tensorMap)["az"]);

    CuMatrix *adj =
        (CuMatrix *)gpuComm->engine->adjIn;  // any engineer with pursuit should
                                             // not write this;too ugly
    CuMatrix e = *adj;
    unsigned edgCnt = e.nnz;

    CuMatrix az_edge=cu.wrapMatrix(Matrix(e.nnz,1,(char*)NULL));
    auto cusparseStat = hipsparseSgthr(
        cu.spHandle, e.nnz, zaTensor.devPtr, az_edge.devPtr,
        e.csrRowInd,  // Not sure need to see the actually adjmatrix***
        HIPSPARSE_INDEX_BASE_ZERO);  // gather the 1st half of az//
    assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);

    CuMatrix d_lrelu_edge = cu.leakyReluPrime(az_edge, 0.01);  // nnz x 1
    az_edge.explicitFree();

    // cout << "gatherRows d_P_edge \n";
    auto gradTensor = cu.wrapMatrix((*gpuComm->tensorMap)["grad"]);
    // cout << "gradTensor.shape " << gradTensor.shape() << endl;
    // cout << "e.nnz " << e.nnz << endl;
    auto d_P_edge = cu.gatherRowsGthr(gradTensor, e.csrRowInd, e.nnz);
    gradTensor.explicitFree();

    // cout << "scaleRowsByVector\n";
    cu.scaleRowsByVector(d_P_edge, d_lrelu_edge);  //(|E|, featDim)
    auto d_Act = d_P_edge;
    d_lrelu_edge.explicitFree();
    // cout << "d_Act.shape() " << d_Act.shape() << endl

    if (layer != 0) {
        CuMatrix a = cu.wrapMatrix(msgService.getaMatrix(layer));
        // Shape of dA: (|E|, 1), serve as gradient of each edge for backward
        // agg
        auto dA = d_Act.dot(a);
        dA.setData((*gpuComm->tensorMap)["dA"].getData());
        dA.updateMatrixFromGPU();
        dA.explicitFree();
    }
    cout << "d_Act_reduce\n";
    auto d_Act_reduce = cu.reduceColumns(d_Act);
    d_Act.explicitFree();

    cout << "gatherRows gatherRows\n";
    auto z = cu.wrapMatrix((*gpuComm->tensorMap)["z"]);
    cout << "zz=z.dot(z\n";
    auto zz = z.dot(z, true, false);
    cout << "da\n";
    CuMatrix da = zz.dot(d_Act_reduce, false, true);
    da.updateMatrixFromGPU();
    msgService.sendaUpdate(da, layer);
    CuMatrix::freeGPU();
}
void ComputingServer::vtxNNBackward(unsigned layer) {
    Matrix host_h = layer == 0
                        ? (*gpuComm->tensorMap)["h"]
                        : gpuComm->engine->savedNNTensors[layer - 1]["ah"];

    auto weight = cu.wrapMatrix(msgService.getWeightMatrix(layer));
    auto grad = cu.wrapMatrix((*gpuComm->tensorMap)["aTg"]);
    auto h = cu.wrapMatrix(host_h);
    auto weightUpdates = h.dot(grad, true, false);
    cout<<"weightUpdates "<<weightUpdates.shape()<<endl;
    weightUpdates.updateMatrixFromGPU();
    msgService.sendWeightUpdate(weightUpdates, layer);
    weightUpdates.free();

    if (layer != 0) {
        auto resultGrad = grad.dot(weight, false, true);
        resultGrad.setData(
            gpuComm->engine->savedNNTensors[layer - 1]["grad"].getData());
        resultGrad.updateMatrixFromGPU();
        printLog(
            nodeId, "layer %u, resultG %s, output %s", layer,
            resultGrad.shape().c_str(),
            gpuComm->engine->savedNNTensors[layer - 1]["grad"].shape().c_str());
    }

    CuMatrix::freeGPU();
}

// end GAT

void ComputingServer::processForward(unsigned layer, bool lastLayer) {
    if (layer == 0) CuMatrix::freeGPU();

    Matrix feats = (*gpuComm->tensorMap)["ah"];
    Matrix weight = msgService.getWeightMatrix(layer);
    CuMatrix z = cu.dot(feats, weight);

    if (!lastLayer) {
        Matrix savedTensor = (*gpuComm->tensorMap)["z"];
        Matrix outputTensor = (*gpuComm->tensorMap)["h"];
        FeatType *act_z = outputTensor.getData();
        FeatType *z_data = savedTensor.getData();
        memcpy(z_data, z.getData(), z.getDataSize());
        cu.activate(z);  // z data get activated ...
        z.updateMatrixFromGPU();
        memcpy(act_z, z.getData(), z.getDataSize());

    } else {  // do the last layer + the bp with it
        CuMatrix cuPredictions = cu.softmaxRows(z);
        gradLoss(layer, cuPredictions);
    }
    delete[] z.getData();
}

void ComputingServer::processBackward(unsigned layer) {
    gradLayer(layer);
    if (layer == 0) msgService.prefetchWeightsMatrix(totalLayers);
}

void ComputingServer::gradLayer(unsigned layer) {
    Matrix grad = (*gpuComm->tensorMap)["aTg"];
    CuMatrix cuGrad = cu.wrapMatrix(grad);
    Matrix z = (*gpuComm->tensorMap)["z"];
    CuMatrix cuZ = cu.wrapMatrix(z);
    Matrix ah = (*gpuComm->tensorMap)["ah"];
    CuMatrix cuAh = cu.wrapMatrix(ah);

    CuMatrix interGrad = cu.activateBackward(cuAh, cuZ, cuGrad);
    CuMatrix cuWeightUpdates = cuAh.dot(interGrad, true, false);

    Matrix weightUpdates = cuWeightUpdates.getMatrix();
    Matrix weight = msgService.getWeightMatrix(layer);
    CuMatrix cuWeights = cu.wrapMatrix(weight);
    if (layer != 0) {
        CuMatrix resultGrad = interGrad.dot(cuWeights, false, true);
        resultGrad.setData((*gpuComm->tensorMap)["grad"].getData());
        resultGrad.updateMatrixFromGPU();
    }

    msgService.sendWeightUpdate(weightUpdates, layer);
}

void ComputingServer::gradLoss(unsigned layer, CuMatrix pred, bool report) {
    // here it can be optimized by fetching directly from Forward;
    Matrix labels = (*gpuComm->tensorMap)["lab"];
    CuMatrix cuLabels = cu.wrapMatrix(labels);
    CuMatrix d_output = cu.hadamardSub(pred, cuLabels);

    if (report) {
        float acc, loss;
        cu.getTrainStat(pred, cuLabels, acc, loss);
        printLog(nodeId, "batch Acc: %f, Loss: %f\n", acc, loss);
    }

    Matrix weight = msgService.getWeightMatrix(layer);
    CuMatrix cuWeights = cu.wrapMatrix(weight);
    CuMatrix interGrad = d_output.dot(cuWeights, false, true);
    interGrad.setData((*gpuComm->tensorMap)["grad"].getData());
    interGrad.updateMatrixFromGPU();

    Matrix ah = (*gpuComm->tensorMap)["ah"];
    CuMatrix cuAh = cu.wrapMatrix(ah);
    CuMatrix cuWeightUpdates = cuAh.dot(d_output, true, false);
    Matrix weightUpdates = cuWeightUpdates.getMatrix();
    msgService.sendWeightUpdate(weightUpdates, layer);
}
