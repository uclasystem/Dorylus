#include "cu_matrix.cuh"

std::set<FeatType *> CuMatrix::MemoryPool;
CuMatrix::CuMatrix(Matrix M, const hipblasHandle_t &handle_)
    : Matrix(M.getRows(), M.getCols(), M.getData()) {
    cudaStat = hipError_t();
    handle = handle_;
    nnz = 0;
    csrVal = NULL;
    csrColInd = NULL;
    csrRowInd = NULL;
    isSparse = 0;
    deviceMalloc();
    if (getData() != NULL) deviceSetMatrix();
}

void CuMatrix::explicitFree() {
    CuMatrix::MemoryPool.erase(devPtr);
    hipFree(devPtr);
}

Matrix CuMatrix::getMatrix() {
    updateMatrixFromGPU();
    return Matrix(getRows(), getCols(), getData());
}

void CuMatrix::freeGPU() {
    for (auto ptr : MemoryPool) hipFree(ptr);
}

void CuMatrix::loadSpCSR(hipsparseHandle_t &handle, Graph &graph) {
    unsigned total = graph.dstGhostCnt + graph.localVtxCnt;
    isSparse = true;
    nnz = graph.backwardAdj.nnz;

    cudaStat = hipMalloc((void **)&csrVal, nnz * sizeof(EdgeType));
    assert(cudaStat == hipSuccess);
    cudaStat = hipMalloc((void **)&csrColInd, nnz * sizeof(unsigned));
    assert(cudaStat == hipSuccess);
    cudaStat = hipMalloc((void **)&csrRowInd, nnz * sizeof(unsigned));
    assert(cudaStat == hipSuccess);
    cudaStat = hipMalloc((void **)&csrRowPtr,
                          (graph.localVtxCnt + 1) * sizeof(unsigned));
    assert(cudaStat == hipSuccess);
    cudaStat = hipMemcpy(csrVal, graph.backwardAdj.values,
                          sizeof(EdgeType) * nnz, hipMemcpyHostToDevice);
    assert(cudaStat == hipSuccess);
    cudaStat = hipMemcpy(csrColInd, graph.backwardAdj.columnIdxs,
                          sizeof(unsigned) * nnz, hipMemcpyHostToDevice);
    assert(cudaStat == hipSuccess);
    unsigned *rowPtrs = new unsigned[graph.localVtxCnt + 1];
    for (unsigned i = 0; i < graph.localVtxCnt + 1; ++i) {
        rowPtrs[i] = (unsigned)(graph.backwardAdj.rowPtrs[i]);
    }
    cudaStat = hipMemcpy(csrRowPtr, rowPtrs,
                          sizeof(unsigned) * (graph.localVtxCnt + 1),
                          hipMemcpyHostToDevice);
    assert(cudaStat == hipSuccess);

    setRows(graph.localVtxCnt);
    setCols(total);

    hipsparseXcsr2coo(handle, csrRowPtr, nnz, getCols(), csrRowInd,
                     HIPSPARSE_INDEX_BASE_ZERO);
}

void CuMatrix::loadSpCSC(hipsparseHandle_t &handle, Graph &graph) {
    unsigned total = graph.srcGhostCnt + graph.localVtxCnt;
    isSparse = true;
    nnz = graph.forwardAdj.nnz;

    cudaStat = hipMalloc((void **)&csrVal, nnz * sizeof(EdgeType));
    assert(cudaStat == hipSuccess);
    cudaStat = hipMalloc((void **)&csrColInd, nnz * sizeof(unsigned));
    assert(cudaStat == hipSuccess);
    cudaStat = hipMalloc((void **)&csrRowInd, nnz * sizeof(unsigned));
    assert(cudaStat == hipSuccess);
    cudaStat = hipMalloc((void **)&csrRowPtr,
                          (graph.localVtxCnt + 1) * sizeof(unsigned));
    assert(cudaStat == hipSuccess);
    cudaStat = hipMemcpy(csrVal, graph.forwardAdj.values,
                          sizeof(EdgeType) * nnz, hipMemcpyHostToDevice);
    assert(cudaStat == hipSuccess);
    cudaStat = hipMemcpy(csrColInd, graph.forwardAdj.rowIdxs,
                          sizeof(unsigned) * nnz, hipMemcpyHostToDevice);
    assert(cudaStat == hipSuccess);
    unsigned *columnPtrs = new unsigned[graph.localVtxCnt + 1];
    for (unsigned i = 0; i < graph.localVtxCnt + 1; ++i) {
        columnPtrs[i] = (unsigned)(graph.forwardAdj.columnPtrs[i]);
    }
    cudaStat = hipMemcpy(csrRowPtr, columnPtrs,
                          sizeof(unsigned) * (graph.localVtxCnt + 1),
                          hipMemcpyHostToDevice);
    assert(cudaStat == hipSuccess);

    setRows(graph.localVtxCnt);
    setCols(total);

    hipsparseXcsr2coo(handle, csrRowPtr, nnz, getRows(), csrRowInd,
                     HIPSPARSE_INDEX_BASE_ZERO);
}
// You could probably make this function load two matrices instead of pointers
// and numbers
void CuMatrix::loadSpDense(FeatType *vtcsTensor, FeatType *ghostTensor,
                           unsigned numLocalVertices, unsigned numGhostVertices,
                           unsigned numFeat) {
    // Still row major
    unsigned totalVertices = (numLocalVertices + numGhostVertices);
    cudaStat = hipMalloc((void **)&devPtr,
                          numFeat * sizeof(FeatType) * totalVertices);
    assert(cudaStat == hipSuccess);
    cudaStat = hipMemcpy(devPtr, vtcsTensor,
                          sizeof(FeatType) * numLocalVertices * numFeat,
                          hipMemcpyHostToDevice);
    assert(cudaStat == hipSuccess);
    cudaStat = hipMemcpy(devPtr + numLocalVertices * numFeat, ghostTensor,
                          sizeof(FeatType) * numGhostVertices * numFeat,
                          hipMemcpyHostToDevice);
    assert(cudaStat == hipSuccess);
    setRows(totalVertices);
    setCols(numFeat);
    MemoryPool.insert(devPtr);
}

CuMatrix CuMatrix::extractRow(unsigned row) {
    FeatType *data = getData() ? (getData() + row * getCols()) : NULL;
    CuMatrix rowVec;
    rowVec.handle = handle;
    rowVec.setData(data);
    rowVec.setRows(1);
    rowVec.setCols(getCols());
    rowVec.devPtr = devPtr + row * getCols();
    return rowVec;
}

void CuMatrix::deviceMalloc() {
    unsigned rows = this->getRows();
    unsigned cols = this->getCols();
    cudaStat = hipMalloc((void **)&devPtr, rows * cols * sizeof(FeatType));
    if (cudaStat != hipSuccess) {
        printf("device memory allocation failed %u\n", cudaStat);
        exit(EXIT_FAILURE);
    }
    MemoryPool.insert(devPtr);
}

void CuMatrix::deviceSetMatrix() {
    unsigned rows = this->getRows();
    unsigned cols = this->getCols();
    FeatType *data = this->getData();

    stat = hipblasSetMatrix(rows, cols, sizeof(float), data, rows, devPtr, rows);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        switch (stat) {
            case HIPBLAS_STATUS_NOT_INITIALIZED:
                printf("HIPBLAS_STATUS_NOT_INITIALIZED\n");
                break;
            case HIPBLAS_STATUS_INVALID_VALUE:
                printf("HIPBLAS_STATUS_INVALID_VALUE\n");
                break;
            case HIPBLAS_STATUS_MAPPING_ERROR:
                printf("HIPBLAS_STATUS_MAPPING_ERROR\n");
                break;
        }
        hipFree(devPtr);
        hipblasDestroy(handle);
        exit(EXIT_FAILURE);
    }
}

void CuMatrix::updateMatrixFromGPU() {
    unsigned rows = this->getRows();
    unsigned cols = this->getCols();
    if (getData() == NULL) setData(new FeatType[getNumElemts()]);
    FeatType *data = this->getData();
    stat = hipblasGetMatrix(rows, cols, sizeof(float), devPtr, rows, data, rows);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("data upload failed\n");
        switch (stat) {
            case HIPBLAS_STATUS_NOT_INITIALIZED:
                printf("HIPBLAS_STATUS_NOT_INITIALIZED\n");
                break;
            case HIPBLAS_STATUS_INVALID_VALUE:
                printf("HIPBLAS_STATUS_INVALID_VALUE\n");
                break;
            case HIPBLAS_STATUS_MAPPING_ERROR:
                printf("HIPBLAS_STATUS_MAPPING_ERROR\n");
                break;
        }
        hipFree(devPtr);
        hipblasDestroy(handle);
        exit(EXIT_FAILURE);
    }
}

CuMatrix::~CuMatrix() {}

void CuMatrix::scale(const float &alpha) {
    hipblasSscal(handle, getNumElemts(), &alpha, devPtr, 1);
}

// definitely should have gemm and gemv handled differently
CuMatrix CuMatrix::dot(CuMatrix &B, bool A_trans, bool B_trans, float alpha,
                       float beta) {
    if (handle != B.handle) {
        std::cout << "Handle don't match\n";
        exit(EXIT_FAILURE);
    }
    hipblasOperation_t ATrans = A_trans ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t BTrans = B_trans ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    // 1. cublas is using col-major
    // 2. when cpy into/out device memory, it will do Transpose
    // 3. C=AB and C^T= (B^T*A^T)
    // This means just swap the order of multiplicaiton
    // Guide: https://peterwittek.com/cublas-matrix-c-style.html
    Matrix AT = Matrix(getCols(), getRows(), getData());
    Matrix BT = Matrix(B.getCols(), B.getRows(), B.getData());

    unsigned CRow = A_trans ? AT.getRows() : getRows();
    unsigned CCol = B_trans ? BT.getCols() : B.getCols();
    Matrix mat_C(CRow, CCol, (char *)NULL);  // real C

    unsigned k = A_trans ? getRows() : getCols();
    CuMatrix C(mat_C, handle);

    stat = hipblasSgemm(handle, BTrans, ATrans, C.getCols(), C.getRows(), k,
                       &alpha, B.devPtr, B.getCols(), devPtr, getCols(), &beta,
                       C.devPtr, C.getCols());
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("SGEMM ERROR\n");
        hipFree(devPtr);
        hipblasDestroy(handle);
        exit(EXIT_FAILURE);
    }
    return C;
}
//Fake inplace actually really slow...
CuMatrix CuMatrix::transpose(bool inplace) {
    if (inplace) {
        if (getData() == NULL) 
            updateMatrixFromGPU();
        explicitFree();
        Matrix result = transpose_();
        return CuMatrix(result, handle);
    } else {
        CuMatrix res(Matrix(getCols(), getRows(), (FeatType *)NULL), handle);

        float alpha = 1.0;
        float beta = 0.;

        stat = hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, getRows(),
                           getCols(), &alpha, devPtr, getCols(), &beta, devPtr,
                           getRows(), res.devPtr, getRows());
        if (stat != HIPBLAS_STATUS_SUCCESS) {
            hipblasDestroy(handle);
            exit(EXIT_FAILURE);
        }

        return res;
    }
}
