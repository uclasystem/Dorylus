#include "hip/hip_runtime.h"
#include "comp_unit.cuh"
#include "cuda_ops.cuh"

#define ACTIVATION HIPDNN_ACTIVATION_TANH

const float alpha = 1.0f, beta = 0.0f;
using namespace std;
void cudaErrCheck(hipError_t stat) {
    if (stat != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(stat));
        fprintf(stderr, "CUDA Error: %d\n", (stat));
    }
}

ComputingUnit *ComputingUnit::instance = nullptr;
ComputingUnit &ComputingUnit::getInstance() {
    if (instance == nullptr) instance = new ComputingUnit();
    return *instance;
}

ComputingUnit::ComputingUnit() {
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("CUBLAS initialization failed\n");
        printf("CUBLAS stat %u\n", stat);
        exit(EXIT_FAILURE);
    }
    hipdnnStatus_t err = hipdnnCreate(&cudnnHandle);
    if (err != HIPDNN_STATUS_SUCCESS) {
        std::cout << "Error occurred: " << err << std::endl;
        std::exit(EXIT_FAILURE);
    }
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
    auto cusparseStat = hipsparseCreate(&spHandle);
    assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);
}

CuMatrix ComputingUnit::wrapMatrix(Matrix m) { return CuMatrix(m, handle); }

CuMatrix ComputingUnit::aggregate(CuMatrix &sparse, CuMatrix &dense,
                                  CuMatrix &norms) {
    CuMatrix C(Matrix(dense.getCols(), sparse.getRows(), (FeatType *)NULL),
               handle);

    hipsparseSpMatDescr_t desA;
    hipsparseDnMatDescr_t desB;
    hipsparseDnMatDescr_t desC;

    const float agg_beta = 0;
    auto cusparseStat = hipsparseCreateCsr(
        &desA, sparse.getRows(), sparse.getCols(), sparse.nnz, sparse.csrRowPtr,
        sparse.csrColInd, sparse.csrVal, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);
    cusparseStat = hipsparseCreateDnMat(&desB, dense.getCols(), dense.getRows(),
                                       dense.getCols(), dense.devPtr,
                                       HIP_R_32F, HIPSPARSE_ORDER_COL);
    assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);
    cusparseStat = hipsparseCreateDnMat(&desC, sparse.getRows(), dense.getCols(),
                                       sparse.getRows(), C.devPtr, HIP_R_32F,
                                       HIPSPARSE_ORDER_COL);
    assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);

    std::size_t buffer_size;
    cusparseStat = hipsparseSpMM_bufferSize(
        spHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_TRANSPOSE, &alpha, desA, desB, &agg_beta, desC,
        HIP_R_32F, HIPSPARSE_MM_ALG_DEFAULT, &buffer_size);
    assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);
    float *buffer;
    cudaErrCheck(hipMalloc((void **)&buffer, buffer_size * sizeof(float)));
    cusparseStat = hipsparseSpMM(spHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                HIPSPARSE_OPERATION_TRANSPOSE, &alpha, desA,
                                desB, &agg_beta, desC, HIP_R_32F,
                                HIPSPARSE_MM_ALG_DEFAULT, buffer);
    assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);
    C = C.transpose();
    scaleRowsByVector(dense, norms);

    hadamardAdd(C, dense);
    hipDeviceSynchronize();
    return C;
}
// This function will scale first nth rows of M based on the length of cuV
void ComputingUnit::scaleRowsByVector(CuMatrix &cuM, CuMatrix &cuV) {
    stat = hipblasSdgmm(handle, HIPBLAS_SIDE_RIGHT, cuM.getCols(), cuV.getRows(),
                       cuM.devPtr, cuM.getCols(), cuV.devPtr, 1, cuM.devPtr,
                       cuM.getCols());
    assert(stat == HIPBLAS_STATUS_SUCCESS);
}

void ComputingUnit::hadamardAdd(CuMatrix &matLeft, CuMatrix &matRight) {
    thrust::device_ptr<float> cuLeft_ptr(matLeft.devPtr);
    thrust::device_ptr<float> cuRight_ptr(matRight.devPtr);
    thrust::transform(cuLeft_ptr, cuLeft_ptr + matLeft.getNumElemts(),
                      cuRight_ptr, cuLeft_ptr, thrust::plus<float>());
}

CuMatrix ComputingUnit::hadamardSub(CuMatrix &matLeft, CuMatrix &matRight) {
    assert(matLeft.getRows() == matRight.getRows());
    assert(matLeft.getCols() == matRight.getCols());
    CuMatrix res(Matrix(matLeft.getRows(), matLeft.getCols(), (FeatType *)NULL),
                 handle);

    thrust::device_ptr<float> cuLeft_ptr(matLeft.devPtr);
    thrust::device_ptr<float> cuRight_ptr(matRight.devPtr);
    thrust::device_ptr<float> res_ptr(res.devPtr);
    thrust::transform(cuLeft_ptr, cuLeft_ptr + matLeft.getNumElemts(),
                      cuRight_ptr, res_ptr, thrust::minus<float>());
    return res;
}

CuMatrix ComputingUnit::hadamardMul(CuMatrix &matLeft, CuMatrix &matRight) {
    assert(matLeft.getRows() == matRight.getRows());
    assert(matLeft.getCols() == matRight.getCols());
    CuMatrix res(Matrix(matLeft.getRows(), matLeft.getCols(), (FeatType *)NULL),
                 handle);

    thrust::device_ptr<float> cuLeft_ptr(matLeft.devPtr);
    thrust::device_ptr<float> cuRight_ptr(matRight.devPtr);
    thrust::device_ptr<float> res_ptr(res.devPtr);

    thrust::transform(cuLeft_ptr, cuLeft_ptr + matLeft.getNumElemts(),
                      cuRight_ptr, res_ptr, thrust::multiplies<float>());

    return res;
}

CuMatrix ComputingUnit::softmaxRows(CuMatrix &mat) {
    CuMatrix res(Matrix(mat.getRows(), mat.getCols(), (FeatType *)NULL),
                 handle);
    hipdnnTensorDescriptor_t srcTensorDesc, sftTensorDesc;
    hipdnnCreateTensorDescriptor(&srcTensorDesc);
    hipdnnCreateTensorDescriptor(&sftTensorDesc);
    hipdnnSetTensor4dDescriptor(srcTensorDesc, HIPDNN_TENSOR_NCHW,
                               HIPDNN_DATA_FLOAT, mat.getRows(), 1, 1,
                               mat.getCols());
    hipdnnSetTensor4dDescriptor(sftTensorDesc, HIPDNN_TENSOR_NCHW,
                               HIPDNN_DATA_FLOAT, mat.getRows(), 1, 1,
                               mat.getCols());
    hipdnnSoftmaxForward(cudnnHandle, HIPDNN_SOFTMAX_ACCURATE,
                        HIPDNN_SOFTMAX_MODE_INSTANCE, &alpha, srcTensorDesc,
                        mat.devPtr, &beta, sftTensorDesc, res.devPtr);
    return res;
}

CuMatrix ComputingUnit::activateBackward(CuMatrix &x, CuMatrix &y,
                                         CuMatrix &dy) {
    FeatType *x_d = new FeatType[y.getNumElemts()];
    FeatType *dx_d = new FeatType[y.getNumElemts()];
    memset(dx_d, 0, y.getDataSize());
    memset(x_d, 0, y.getDataSize());
    CuMatrix dx(Matrix(y.getRows(), y.getCols(), dx_d), handle);
    CuMatrix x_(Matrix(y.getRows(), y.getCols(), x_d), handle);

    hipdnnActivationDescriptor_t actDesc;
    hipdnnCreateActivationDescriptor(&actDesc);
    hipdnnSetActivationDescriptor(actDesc, ACTIVATION, HIPDNN_NOT_PROPAGATE_NAN,
                                 0.0);
    hipdnnTensorDescriptor_t yDesc;
    hipdnnCreateTensorDescriptor(&yDesc);
    hipdnnSetTensor4dDescriptor(yDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                               y.getRows(), 1, 1, y.getCols());
    auto error = hipdnnActivationBackward(cudnnHandle, actDesc, &alpha, yDesc,
                                         y.devPtr, yDesc, dy.devPtr, yDesc,
                                         x_.devPtr, &beta, yDesc, dx.devPtr);
    assert(HIPDNN_STATUS_SUCCESS == error);
    delete[] x_d;
    delete[] dx_d;

    return dx;
}

CuMatrix ComputingUnit::dot(Matrix &A, Matrix &B) {
    CuMatrix devA(A, handle);
    CuMatrix devB(B, handle);
    CuMatrix devC = devA.dot(devB);
    devC.updateMatrixFromGPU();
    return devC;
}

void ComputingUnit::activate(CuMatrix &A) {
    hipdnnTensorDescriptor_t srcTensorDesc;
    hipdnnCreateTensorDescriptor(&srcTensorDesc);
    hipdnnSetTensor4dDescriptor(srcTensorDesc, HIPDNN_TENSOR_NCHW,
                               HIPDNN_DATA_FLOAT, A.getRows(), 1, 1,
                               A.getCols());

    hipdnnActivationDescriptor_t actDesc;
    hipdnnCreateActivationDescriptor(&actDesc);
    hipdnnSetActivationDescriptor(actDesc, ACTIVATION, HIPDNN_PROPAGATE_NAN, 0.0);
    hipdnnActivationForward(cudnnHandle, actDesc, &alpha, srcTensorDesc,
                           A.devPtr, &beta, srcTensorDesc, A.devPtr);
}

//** much slower than CPU only if Input Matrices are not loaded in GPU
// beforehand
unsigned ComputingUnit::checkAccuracy(CuMatrix &predictions, CuMatrix &labels) {
    unsigned rowSize = predictions.getCols();

    thrust::device_vector<FeatType *> row_starts(predictions.getRows());
    thrust::counting_iterator<int> idxfirst(0);

    thrust::transform(idxfirst, idxfirst + predictions.getRows(),
                      row_starts.begin(),
                      setRowStarts(predictions.devPtr, rowSize));
    thrust::device_vector<unsigned> pred_results(predictions.getRows());
    thrust::transform(row_starts.begin(), row_starts.end(),
                      pred_results.begin(), findRowMaximum(rowSize));

    thrust::transform(idxfirst, idxfirst + predictions.getRows(),
                      row_starts.begin(), setRowStarts(labels.devPtr, rowSize));
    thrust::device_vector<unsigned> true_results(predictions.getRows());
    thrust::transform(pred_results.begin(), pred_results.end(),
                      row_starts.begin(), true_results.begin(),
                      isPredictCorrect(rowSize));

    unsigned totalCorrect =
        thrust::reduce(true_results.begin(), true_results.end(), (unsigned)0,
                       thrust::plus<unsigned>());
    return totalCorrect;
}

//** much slower than CPU only if Input Matrices are not loaded in GPU
// beforehand
float ComputingUnit::checkLoss(CuMatrix &preds, CuMatrix &labels) {
    unsigned rowSize = preds.getCols();

    thrust::counting_iterator<int> idxfirst(0);
    thrust::device_vector<FeatType *> row_starts(preds.getRows());
    thrust::transform(idxfirst, idxfirst + preds.getRows(), row_starts.begin(),
                      setRowStarts(labels.devPtr, rowSize));
    thrust::device_vector<unsigned> true_labels(preds.getRows());
    thrust::transform(row_starts.begin(), row_starts.end(), true_labels.begin(),
                      findTrueLabel(rowSize));
    thrust::transform(idxfirst, idxfirst + preds.getRows(), row_starts.begin(),
                      setRowStarts(preds.devPtr, rowSize));
    thrust::device_vector<FeatType> losses(preds.getRows());
    thrust::transform(true_labels.begin(), true_labels.end(),
                      row_starts.begin(), losses.begin(), getLoss(rowSize));
    float totalLoss = thrust::reduce(losses.begin(), losses.end(), (float)0,
                                     thrust::plus<float>());
    return totalLoss;
}

void ComputingUnit::getTrainStat(CuMatrix &preds, CuMatrix &labels, float &acc,
                                 float &loss) {
    loss = checkLoss(preds, labels) / labels.getRows();
    acc = checkAccuracy(preds, labels) / (float)labels.getRows();
    // float * l = new float [labels.getNumElemts()];
    // float * p = new float [preds.getNumElemts()];
    // preds.setData(p);
    // labels.setData(l);
    // preds.updateMatrixFromGPU();
    // labels.updateMatrixFromGPU();
    // acc = 0.0;
    // loss = 0.0;
    // unsigned featDim=labels.getCols();
    // for (unsigned i = 0; i < labels.getRows(); i++) {
    //     FeatType *currLabel = l + i * labels.getCols();
    //     FeatType *currPred = p + i * labels.getCols();
    //     acc += currLabel[argmax(currPred, currPred + featDim)];
    //     loss -= std::log(currPred[argmax(currLabel, currLabel + featDim)]);
    // }
    // acc /= labels.getRows();
    // loss /= labels.getRows();
    // printLog(getNodeId(), "batch loss %f, batch acc %f", loss, acc);
}